#include <iostream>
#include "hip/hip_runtime.h"

#include <chrono>
#define ARRAY_SIZE 10000
#define NUM_THREADS 256

__global__ void kernel(int* array, int N) {
    int index_x = threadIdx.x + blockIdx.x * blockDim.x;
    int stride_x = blockDim.x * gridDim.x;
    
    for(int i = index_x; i < N; i += stride_x){
        array[i] = i;
    }
}

__global__ void perfBugged(int* array) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < ARRAY_SIZE) {
        // Poor cache utilization: Non-contiguous memory access
        array[tid] = tid;
    }
}


int main() {
    int* d_array, * d_array_2;
    hipMalloc((void**)&d_array, ARRAY_SIZE * sizeof(int));
    hipMalloc((void**)&d_array_2, ARRAY_SIZE * sizeof(int));
    int num_blocks = (ARRAY_SIZE + NUM_THREADS - 1) / NUM_THREADS;

    auto start = std::chrono::high_resolution_clock::now();
    perfBugged<<<(ARRAY_SIZE + NUM_THREADS - 1) / NUM_THREADS, NUM_THREADS>>>(d_array);
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Performance Bugged Kernel execution time: " << duration.count() << " seconds" << std::endl;

    start = std::chrono::high_resolution_clock::now();
    kernel<<<num_blocks, NUM_THREADS>>>(d_array, ARRAY_SIZE);
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "LLM Fixed Kernel execution time: " << duration.count() << " seconds" << std::endl;


    hipFree(d_array); hipFree(d_array_2);

    return 0;
}
