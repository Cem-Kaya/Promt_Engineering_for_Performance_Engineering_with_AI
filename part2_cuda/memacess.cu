#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <chrono>
#define MATRIX_SIZE 4096

// Matrix allocated in global shared memory
__global__ void sharedMemoryKernel(float* matrix) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float value = matrix[tid];
    value += 1.0f;
    // Perform computations using the shared matrix...
}

// Matrix allocated in constant memory
__constant__ float constMatrix[MATRIX_SIZE][MATRIX_SIZE];

__global__ void constantMemoryKernel() {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float value = constMatrix[threadIdx.y][threadIdx.x];
    value += 1.0f;
    // Perform computations using the constant matrix...
}

int main() {
    // Initialize matrix
    float h_matrix[MATRIX_SIZE][MATRIX_SIZE];
    for (int i = 0; i < MATRIX_SIZE; ++i) {
        for (int j = 0; j < MATRIX_SIZE; ++j) {
            h_matrix[i][j] = static_cast<float>(i * MATRIX_SIZE + j);
        }
    }

    float* d_matrix;
    hipMalloc((void**)&d_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    hipMemcpy(d_matrix, h_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);
    
    hipEvent_t start, stop;
    float elapsed1, elapsed2;
s
    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    sharedMemoryKernel<<<MATRIX_SIZE, MATRIX_SIZE>>>(d_matrix);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed1, start, stop);
    hipDeviceSynchronize();
    printf("Time to calculate results on GPU: %f ms\n", elapsed1);

    hipMemcpyToSymbol(HIP_SYMBOL(constMatrix), h_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));

    // Launch constant memory kernel
    hipEventRecord(start, 0);

    constantMemoryKernel<<<MATRIX_SIZE, MATRIX_SIZE>>>();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed2, start, stop);
    printf("Time to calculate results on GPU: %f ms\n", elapsed2);

    hipDeviceSynchronize();
    hipFree(d_matrix);

    return 0;
}
