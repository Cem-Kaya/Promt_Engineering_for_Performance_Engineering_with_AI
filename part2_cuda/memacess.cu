#include <iostream>
#include "hip/hip_runtime.h"

#include <chrono>
#define MATRIX_SIZE 16

// Matrix allocated in global shared memory
__global__ void sharedMemoryKernel(float* matrix) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float value = matrix[tid];
    value += 1.0f;
    // Perform computations using the shared matrix...
}

// Matrix allocated in constant memory
__constant__ float constMatrix[MATRIX_SIZE][MATRIX_SIZE];

__global__ void constantMemoryKernel() {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    float value = constMatrix[threadIdx.y][threadIdx.x];
    value += 1.0f;
    // Perform computations using the constant matrix...
}

int main() {
    // Initialize matrix
    float h_matrix[MATRIX_SIZE][MATRIX_SIZE];
    for (int i = 0; i < MATRIX_SIZE; ++i) {
        for (int j = 0; j < MATRIX_SIZE; ++j) {
            h_matrix[i][j] = static_cast<float>(i * MATRIX_SIZE + j);
        }
    }

    float* d_matrix;
    hipMalloc((void**)&d_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));
    hipMemcpy(d_matrix, h_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(float), hipMemcpyHostToDevice);

    sharedMemoryKernel<<<8, 32>>>(d_matrix);
    hipDeviceSynchronize();

    hipMemcpyToSymbol(HIP_SYMBOL(constMatrix), h_matrix, MATRIX_SIZE * MATRIX_SIZE * sizeof(float));

    // Launch constant memory kernel
    constantMemoryKernel<<<8, 32>>>();
    hipDeviceSynchronize();

    hipFree(d_matrix);

    return 0;
}
