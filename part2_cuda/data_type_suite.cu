#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define ARRAY_SIZE 10000000

// CUDA kernel to access array elements with int indices
__global__ void intIndexingKernel(int* array)
{
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    int value = array[index];
}

// CUDA kernel to access array elements with long long indices
__global__ void longLongIndexingKernel(long long* array)
{
    long long index = threadIdx.x + blockIdx.x * blockDim.x;
    long long value = array[index];
}

// CUDA kernel to access array elements with unsigned long long indices
__global__ void unsignedLongLongIndexingKernel(unsigned long long* array)
{
    unsigned long long index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long value = array[index];
}

int main()
{
    int* intArray;
    long long* longLongArray;
    unsigned long long* unsignedLongLongArray;

    hipMalloc((void**)&intArray, ARRAY_SIZE * sizeof(int));
    hipMalloc((void**)&longLongArray, ARRAY_SIZE * sizeof(long long));
    hipMalloc((void**)&unsignedLongLongArray, ARRAY_SIZE * sizeof(unsigned long long));

    hipEvent_t start, stop;
    float elapsed1, elapsed2, elapsed3;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    intIndexingKernel<<<(ARRAY_SIZE + 255) / 256, 256>>>(intArray);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed1, start, stop);
    hipDeviceSynchronize();
    printf("Time to calculate results on GPU: %f ms\n", elapsed1);
    printf("Effective performance: %f GB/s\n", (ARRAY_SIZE * sizeof(int)) / (elapsed1 * 1e6));
    printf("Effective bandwidth: %f GB/s\n", (ARRAY_SIZE * sizeof(int)) / (elapsed1 * 1e6));



    hipEventRecord(start, 0);
    longLongIndexingKernel<<<(ARRAY_SIZE + 255) / 256, 256>>>(longLongArray);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed2, start, stop);
    hipDeviceSynchronize();
    printf("Time to calculate results on GPU: %f ms\n", elapsed2);
    printf("Effective performance: %f GB/s\n", (ARRAY_SIZE * sizeof(long long)) / (elapsed2 * 1e6));
    printf("Effective bandwidth: %f GB/s\n", (ARRAY_SIZE * sizeof(long long)) / (elapsed2 * 1e6));


    hipEventRecord(start, 0);
    unsignedLongLongIndexingKernel<<<(ARRAY_SIZE + 255) / 256, 256>>>(unsignedLongLongArray);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed3, start, stop);
    hipDeviceSynchronize();
    printf("Time to calculate results on GPU: %f ms\n", elapsed3);
    printf("Effective performance: %f GB/s\n", (ARRAY_SIZE * sizeof(unsigned long long)) / (elapsed3 * 1e6));
    printf("Effective bandwidth: %f GB/s\n", (ARRAY_SIZE * sizeof(unsigned long long)) / (elapsed3 * 1e6));


    hipFree(intArray);
    hipFree(longLongArray);
    hipFree(unsignedLongLongArray);

    return 0;
}
