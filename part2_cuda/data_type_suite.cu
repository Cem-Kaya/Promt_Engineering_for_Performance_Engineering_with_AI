#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define ARRAY_SIZE 10000000
/**
 * FILEPATH: /home/cankorkmaz/proj_406/benchmarks/data_type_suite.cu
 * 
 * This code block contains two CUDA kernels to access array elements with unsigned int and unsigned long long indices.
 * Performance of each kernel is measured and printed to the console.
 * 
 * BEGIN: ed8c6549bwf9
 * 
 *  64 bit unsigned long long indexing peformance
 * Time to calculate results on GPU: 0.090048 ms
 * Effective bandwidth: 444.207533 GB/s
 * 
 *  32 bit unsigned integer indexing peformance
 * Time to calculate results on GPU: 0.075520 ms
 * Effective bandwidth: 1059.322015 GB/s
 * 
 * END: ed8c6549bwf9
 */


__global__ void unsignedIntIndexingKernel(unsigned int* array)
{
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < ARRAY_SIZE) {
        unsigned int value = array[index];
    }
}

// CUDA kernel to access array elements with unsigned long long indices
__global__ void unsignedLongLongIndexingKernel(unsigned long long* array)
{
    unsigned long long index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long value = array[index];
}

int main()
{
    unsigned int* unsignedIntArray;

    unsigned long long* unsignedLongLongArray;

    hipMalloc((void**)&unsignedIntArray, ARRAY_SIZE * sizeof(unsigned int));
    hipMalloc((void**)&unsignedLongLongArray, ARRAY_SIZE * sizeof(unsigned long long));

    hipEvent_t start, stop;
    float elapsed1, elapsed3;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    unsignedIntIndexingKernel<<<(ARRAY_SIZE + 255) / 256, 256>>>(unsignedIntArray);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed1, start, stop);
    hipDeviceSynchronize();
    printf("Time to calculate results on GPU: %f ms\n", elapsed1);
    printf("Effective bandwidth: %f GB/s\n", (ARRAY_SIZE * sizeof(int)) / (elapsed1 * 1e6));

    hipEventRecord(start, 0);
    unsignedLongLongIndexingKernel<<<(ARRAY_SIZE + 255) / 256, 256>>>(unsignedLongLongArray);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed3, start, stop);
    hipDeviceSynchronize();
    printf("Time to calculate results on GPU: %f ms\n", elapsed3);
    printf("Effective bandwidth: %f GB/s\n", (ARRAY_SIZE * sizeof(unsigned long long)) / (elapsed3 * 1e6));


    hipFree(unsignedIntArray);
    hipFree(unsignedLongLongArray);

    return 0;
}
