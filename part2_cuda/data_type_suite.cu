#include <iostream>
#include <stdio.h>
#include "hip/hip_runtime.h"

#define ARRAY_SIZE 10000000

__global__ void unsignedIntIndexingKernel(unsigned int* array)
{
    unsigned int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (index < ARRAY_SIZE) {
        unsigned int value = array[index];
    }
}

// CUDA kernel to access array elements with unsigned long long indices
__global__ void unsignedLongLongIndexingKernel(unsigned long long* array)
{
    unsigned long long index = threadIdx.x + blockIdx.x * blockDim.x;
    unsigned long long value = array[index];
}

int main()
{
    unsigned int* unsignedIntArray;

    unsigned long long* unsignedLongLongArray;

    hipMalloc((void**)&unsignedIntArray, ARRAY_SIZE * sizeof(unsigned int));
    hipMalloc((void**)&unsignedLongLongArray, ARRAY_SIZE * sizeof(unsigned long long));

    hipEvent_t start, stop;
    float elapsed1, elapsed3;

    hipEventCreate(&start);
    hipEventCreate(&stop);

    hipEventRecord(start, 0);
    unsignedIntIndexingKernel<<<(ARRAY_SIZE + 255) / 256, 256>>>(unsignedIntArray);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed1, start, stop);
    hipDeviceSynchronize();
    printf("Time to calculate results on GPU: %f ms\n", elapsed1);
    printf("Effective performance: %f GB/s\n", (ARRAY_SIZE * sizeof(int)) / (elapsed1 * 1e6));
    printf("Effective bandwidth: %f GB/s\n", (ARRAY_SIZE * sizeof(int)) / (elapsed1 * 1e6));

    hipEventRecord(start, 0);
    unsignedLongLongIndexingKernel<<<(ARRAY_SIZE + 255) / 256, 256>>>(unsignedLongLongArray);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsed3, start, stop);
    hipDeviceSynchronize();
    printf("Time to calculate results on GPU: %f ms\n", elapsed3);
    printf("Effective performance: %f GB/s\n", (ARRAY_SIZE * sizeof(unsigned long long)) / (elapsed3 * 1e6));
    printf("Effective bandwidth: %f GB/s\n", (ARRAY_SIZE * sizeof(unsigned long long)) / (elapsed3 * 1e6));


    hipFree(unsignedIntArray);
    hipFree(unsignedLongLongArray);

    return 0;
}
