#include <iostream>
#include "hip/hip_runtime.h"

#include <chrono>
#include <stdio.h>


#include <stdio.h>

#include <iostream>
#include <chrono>

#define N 1000
#define BLOCK_SIZE 16

__global__ void matrixMul(int* A, int* B, int* C, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < K && col < K) {
        int sum = 0;

        for (int i = 0; i < K; i++) {
            sum += A[row * K + i] * B[i * K + col];
        }

        C[row * K + col] = sum;
    }
}

__global__ void matrixMulUnrolled(int* A, int* B, int* C, int K) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < K && col < K) {
        int sum = 0;

        for (int i = 0; i < K; i += 4) {
            sum += A[row * K + i] * B[i * K + col] +
                   A[row * K + i + 1] * B[(i + 1) * K + col] +
                   A[row * K + i + 2] * B[(i + 2) * K + col] +
                   A[row * K + i + 3] * B[(i + 3) * K + col];
        }

        C[row * K + col] = sum;
    }
}

int main() {
    // Allocate memory on the host
    int* A = new int[N * N];
    int* B = new int[N * N];
    int* C = new int[N * N];

    // Initialize matrices A and B
    for (int i = 0; i < N * N; i++) {
        A[i] = 1;
        B[i] = 2;
    }

    // Allocate memory on the device
    int* d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, N * N * sizeof(int));
    hipMalloc((void**)&d_B, N * N * sizeof(int));
    hipMalloc((void**)&d_C, N * N * sizeof(int));

    // Copy matrices A and B from host to device
    hipMemcpy(d_A, A, N * N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_B, B, N * N * sizeof(int), hipMemcpyHostToDevice);

    // Set grid and block sizes
    dim3 blockSize(BLOCK_SIZE, BLOCK_SIZE);
    dim3 gridSize((N + blockSize.x - 1) / blockSize.x, (N + blockSize.y - 1) / blockSize.y);

    // Measure execution time using chrono library
    auto start = std::chrono::high_resolution_clock::now();

    // Launch the kernel function
    matrixMul<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);

    // Synchronize to wait for the kernel to finish
    hipDeviceSynchronize();

    auto end = std::chrono::high_resolution_clock::now();
    std::chrono::duration<double> duration = end - start;
    std::cout << "Execution time Normal: " << duration.count() << " seconds" << std::endl;

    start = std::chrono::high_resolution_clock::now();


    // Copy matrix C from device to host
    hipMemcpy(C, d_C, N * N * sizeof(int), hipMemcpyDeviceToHost);
    matrixMulUnrolled<<<gridSize, blockSize>>>(d_A, d_B, d_C, N);  
    hipDeviceSynchronize();
    end = std::chrono::high_resolution_clock::now();
    duration = end - start;
    std::cout << "Execution time Unrolled: " << duration.count() << " seconds" << std::endl;

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] A;
    delete[] B;
    delete[] C;

    return 0;
}
